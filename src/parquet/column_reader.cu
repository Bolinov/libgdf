#include "hip/hip_runtime.h"
/*
 * Copyright 2018 BlazingDB, Inc.
 *     Copyright 2018 Cristhian Alberto Gonzales Castillo <cristhian@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <arrow/util/bit-util.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include "column_reader.h"
#include "dictionary_decoder.cuh"
#include "plain_decoder.cuh"

namespace gdf {
namespace parquet {

#define GDF_DECODER_GPU_VERSION 1 

template <class DataType, class DecoderType>
static inline void
_ConfigureDictionary(
  const ::parquet::Page *                                page,
  std::unordered_map<int, std::shared_ptr<DecoderType>> &decoders,
  const ::parquet::ColumnDescriptor *const               column_descriptor,
  ::arrow::MemoryPool *const                             pool,
  DecoderType **                                         out_decoder) {
    const ::parquet::DictionaryPage *dictionary_page =
      static_cast<const ::parquet::DictionaryPage *>(page);

    int encoding = static_cast<int>(dictionary_page->encoding());
    if (dictionary_page->encoding() == ::parquet::Encoding::PLAIN_DICTIONARY
        || dictionary_page->encoding() == ::parquet::Encoding::PLAIN) {
        encoding = static_cast<int>(::parquet::Encoding::RLE_DICTIONARY);
    }

    auto it = decoders.find(encoding);
    if (it != decoders.end()) {
        throw ::parquet::ParquetException(
          "Column cannot have more than one dictionary.");
    }

    if (dictionary_page->encoding() == ::parquet::Encoding::PLAIN_DICTIONARY
        || dictionary_page->encoding() == ::parquet::Encoding::PLAIN) {
        internal::PlainDecoder<DataType> dictionary(column_descriptor);
        dictionary.SetData(
          dictionary_page->num_values(), page->data(), page->size());

 #ifdef GDF_DECODER_GPU_VERSION
        auto decoder = std::make_shared<internal::DictionaryDecoder<
        DataType, gdf::arrow::internal::RleDecoder> >(column_descriptor, pool);
        decoder->SetDict(&dictionary);
        decoders[encoding] = decoder;
#else
        auto decoder = std::make_shared<internal::DictionaryDecoder<
        DataType, ::arrow::RleDecoder> >(column_descriptor, pool);
        decoder->SetDict(&dictionary);
        decoders[encoding] = decoder;
#endif

    } else {
        ::parquet::ParquetException::NYI(
          "only plain dictionary encoding has been implemented");
    }

    *out_decoder = decoders[encoding].get();
}

static inline bool
_IsDictionaryIndexEncoding(const ::parquet::Encoding::type &e) {
    return e == ::parquet::Encoding::RLE_DICTIONARY
           || e == ::parquet::Encoding::PLAIN_DICTIONARY;
}

template <class DecoderType, class T>
static inline std::int64_t
_ReadValues(DecoderType *decoder, std::int64_t batch_size, T *out) {
    std::int64_t num_decoded =
      decoder->Decode(out, static_cast<int>(batch_size));
    return num_decoded;
}

template <class DataType>
bool
ColumnReader<DataType>::HasNext() {
    if (num_buffered_values_ == 0 || num_decoded_values_ == num_buffered_values_) {
        if (!ReadNewPage() || num_buffered_values_ == 0) { 
            return false; 
        }
    }
    return true;
}

template <class DataType>
bool
ColumnReader<DataType>::ReadNewPage() {
    const std::uint8_t *buffer;

    for (;;) {
        current_page_ = pager_->NextPage();
        if (!current_page_) { return false; }

        if (current_page_->type() == ::parquet::PageType::DICTIONARY_PAGE) {
            _ConfigureDictionary<DataType>(current_page_.get(),
                                           decoders_,
                                           descr_,
                                           pool_,
                                           &current_decoder_);
            continue;
        } else if (current_page_->type() == ::parquet::PageType::DATA_PAGE) {
            const ::parquet::DataPage *page =
              static_cast<const ::parquet::DataPage *>(current_page_.get());

            num_buffered_values_ = page->num_values();
            num_decoded_values_  = 0;
            buffer               = page->data();

            std::int64_t data_size = page->size();

            if (descr_->max_repetition_level() > 0) {
                std::int64_t rep_levels_bytes =
                  repetition_level_decoder_.SetData(
                    page->repetition_level_encoding(),
                    descr_->max_repetition_level(),
                    static_cast<int>(num_buffered_values_),
                    buffer);
                buffer += rep_levels_bytes;
                data_size -= rep_levels_bytes;
            }

            if (descr_->max_definition_level() > 0) {
                std::int64_t def_levels_bytes =
                  definition_level_decoder_.SetData(
                    page->definition_level_encoding(),
                    descr_->max_definition_level(),
                    static_cast<int>(num_buffered_values_),
                    buffer);
                buffer += def_levels_bytes;
                data_size -= def_levels_bytes;
            }

            ::parquet::Encoding::type encoding = page->encoding();

            if (_IsDictionaryIndexEncoding(encoding)) {
                encoding = ::parquet::Encoding::RLE_DICTIONARY;
            }

            auto it = decoders_.find(static_cast<int>(encoding));
            if (it != decoders_.end()) {
                if (encoding == ::parquet::Encoding::RLE_DICTIONARY) {
                    DCHECK(current_decoder_->encoding()
                           == ::parquet::Encoding::RLE_DICTIONARY);
                }
                current_decoder_ = it->second.get();
            } else {
                switch (encoding) {
                case ::parquet::Encoding::PLAIN: {
                    std::shared_ptr<DecoderType> decoder(
                      new internal::PlainDecoder<DataType>(descr_));
                    decoders_[static_cast<int>(encoding)] = decoder;
                    current_decoder_                      = decoder.get();
                    break;
                }
                case ::parquet::Encoding::RLE_DICTIONARY:
                    throw ::parquet::ParquetException(
                      "Dictionary page must be before data page.");

                case ::parquet::Encoding::DELTA_BINARY_PACKED:
                case ::parquet::Encoding::DELTA_LENGTH_BYTE_ARRAY:
                case ::parquet::Encoding::DELTA_BYTE_ARRAY:
                    ::parquet::ParquetException::NYI("Unsupported encoding");

                default:
                    throw ::parquet::ParquetException(
                      "Unknown encoding type.");
                }
            }
            current_decoder_->SetData(static_cast<int>(num_buffered_values_),
                                      buffer,
                                      static_cast<int>(data_size));
            return true;
        } else {
            continue;
        }
    }
    return true;
}

static inline bool
_HasSpacedValues(const ::parquet::ColumnDescriptor *descr) {
    if (descr->max_repetition_level() > 0) {
        return !descr->schema_node()->is_required();
    } else {
        const ::parquet::schema::Node *node = descr->schema_node().get();
        while (node) {
            if (node->is_optional()) { return true; }
            node = node->parent();
        }
        return false;
    }
}

static inline void
_DefinitionLevelsToBitmap(const std::int16_t *def_levels,
                          std::int64_t        num_def_levels,
                          const std::int16_t  max_definition_level,
                          const std::int16_t  max_repetition_level,
                          std::int64_t *      values_read,
                          std::int64_t *      null_count,
                          std::uint8_t *      valid_bits,
                          const std::int64_t  valid_bits_offset) {
    ::arrow::internal::BitmapWriter valid_bits_writer(
      valid_bits, valid_bits_offset, num_def_levels);

    for (std::int64_t i = 0; i < num_def_levels; ++i) {
        if (def_levels[i] == max_definition_level) {
            valid_bits_writer.Set();
        } else if (max_repetition_level > 0) {
            if (def_levels[i] == (max_definition_level - 1)) {
                valid_bits_writer.Clear();
                *null_count += 1;
            } else {
                continue;
            }
        } else {
            if (def_levels[i] < max_definition_level) {
                valid_bits_writer.Clear();
                *null_count += 1;
            } else {
                throw ::parquet::ParquetException(
                  "definition level exceeds maximum");
            }
        }

        valid_bits_writer.Next();
    }
    valid_bits_writer.Finish();
    *values_read = valid_bits_writer.position();
}

template <class DecoderType, class T>
static inline std::int64_t
_ReadValuesSpaced(DecoderType * decoder,
                  std::int64_t  batch_size,
                  T *           out,
                  std::int64_t  null_count,
                  std::uint8_t *valid_bits,
                  std::int64_t  valid_bits_offset) {
    return decoder->DecodeSpaced(out,
                                 static_cast<int>(batch_size),
                                 static_cast<int>(null_count),
                                 valid_bits,
                                 valid_bits_offset);
}

template <typename DataType>
inline std::int64_t
ColumnReader<DataType>::ReadBatchSpaced(std::int64_t  batch_size,
                                        std::int16_t *definition_levels,
                                        std::int16_t *repetition_levels,
                                        T *           values,
                                        std::uint8_t *valid_bits,
                                        std::int64_t  valid_bits_offset, // 
                                        std::int64_t *levels_read,
                                        std::int64_t *values_read,
                                        std::int64_t *nulls_count) {
    if (!HasNext()) {
        *levels_read = 0;
        *values_read = 0;
        *nulls_count = 0;
        return 0;
    }

    std::int64_t total_values;
    //  num_buffered_values_ - num_decoded_values_

    batch_size =
      std::min(batch_size, num_buffered_values_ - num_decoded_values_);

    if (descr_->max_definition_level() > 0) {
        std::int64_t num_def_levels =
          ReadDefinitionLevels(batch_size, definition_levels);

        if (descr_->max_repetition_level() > 0) {
            std::int64_t num_rep_levels =
              ReadRepetitionLevels(batch_size, repetition_levels);
            if (num_def_levels != num_rep_levels) {
                throw ::parquet::ParquetException(
                  "Number of decoded rep / def levels did not match");
            }
        }

        const bool has_spaced_values = _HasSpacedValues(descr_);

        std::int64_t null_count = 0;
        if (!has_spaced_values) {
            int values_to_read = 0;
            for (std::int64_t i = 0; i < num_def_levels; ++i) {
                if (definition_levels[i] == descr_->max_definition_level()) {
                    ++values_to_read;
                }
            }
            std::cout << "*ReadBatchSpaced: before _ReadValues" << std::endl;

            total_values =
              _ReadValues(current_decoder_, values_to_read, values);
            for (std::int64_t i = 0; i < total_values; i++) {
                //check: valid_bits_offset + i
                ::arrow::BitUtil::SetBit(valid_bits, valid_bits_offset + i);
            }
            *values_read = total_values;
        } else {
            std::int16_t max_definition_level = descr_->max_definition_level();
            std::int16_t max_repetition_level = descr_->max_repetition_level();
            _DefinitionLevelsToBitmap(definition_levels,
                                      num_def_levels,
                                      max_definition_level,
                                      max_repetition_level,
                                      values_read,
                                      &null_count,
                                      valid_bits,
                                      valid_bits_offset);
            
            total_values = _ReadValuesSpaced(current_decoder_,
                                             *values_read,
                                             values,
                                             static_cast<int>(null_count),
                                             valid_bits,
                                             valid_bits_offset);
        }
        *levels_read = num_def_levels;
        *nulls_count = null_count;

    } else {
        total_values = _ReadValues(current_decoder_, batch_size, values);
        for (std::int64_t i = 0; i < total_values; i++) {
            ::arrow::BitUtil::SetBit(valid_bits, valid_bits_offset + i);
        }
        *nulls_count = 0;
        *levels_read = total_values;
    }

    ConsumeBufferedValues(*levels_read);

    return total_values;
}


template <class DataType>
inline std::int64_t
ColumnReader<DataType>::ReadBatch(std::int64_t  batch_size,
                                  std::int16_t *def_levels,
                                  std::int16_t *rep_levels,
                                  T *           values,
                                  std::int64_t *values_read) {
    if (!HasNext()) {
        *values_read = 0;
        return 0;
    }
    batch_size = std::min(batch_size, num_buffered_values_ - num_decoded_values_);

    std::int64_t num_def_levels = 0;
    std::int64_t num_rep_levels = 0;

    std::int64_t values_to_read = 0;

    if (descr_->max_definition_level() > 0 && def_levels) {
        num_def_levels = ReadDefinitionLevels(batch_size, def_levels);
        for (std::int64_t i = 0; i < num_def_levels; ++i) {
            if (def_levels[i] == descr_->max_definition_level()) {
                ++values_to_read;
            }
        }
    } else {
        values_to_read = batch_size;
    }

    if (descr_->max_repetition_level() > 0 && rep_levels) {
        num_rep_levels = ReadRepetitionLevels(batch_size, rep_levels);
        if (def_levels && num_def_levels != num_rep_levels) {
            throw ::parquet::ParquetException(
                    "Number of decoded rep / def levels did not match");
        }
    }

    *values_read = _ReadValues(current_decoder_, values_to_read, values);
    std::int64_t total_values = std::max(num_def_levels, *values_read);
    ConsumeBufferedValues(total_values);

    return total_values;
}


template <class DataType>
struct ParquetTraits {};

#define TYPE_TRAITS_FACTORY(ParquetType, GdfDType)                            \
    template <>                                                               \
    struct ParquetTraits<ParquetType> {                                       \
        static constexpr gdf_dtype gdfDType = GdfDType;                       \
    }

TYPE_TRAITS_FACTORY(::parquet::BooleanType, GDF_invalid);
TYPE_TRAITS_FACTORY(::parquet::Int32Type, GDF_INT32);
TYPE_TRAITS_FACTORY(::parquet::Int64Type, GDF_INT64);
// TYPE_TRAITS_FACTORY(::parquet::Int96Type, GDF_invalid);
TYPE_TRAITS_FACTORY(::parquet::FloatType, GDF_FLOAT32);
TYPE_TRAITS_FACTORY(::parquet::DoubleType, GDF_FLOAT64);
// TYPE_TRAITS_FACTORY(::parquet::ByteArrayType, GDF_invalid);
// TYPE_TRAITS_FACTORY(::parquet::FLBAType, GDF_invalid);

#undef TYPE_TRAITS_FACTORY


//#define TO_GDF_COLUMN_USING_READBATCHSPACED 1

#ifdef TO_GDF_COLUMN_USING_READBATCHSPACED
template <class DataType>
std::size_t
ColumnReader<DataType>::ToGdfColumn(std::int16_t *const definition_levels,
                                    std::int16_t *const repetition_levels,
                                    const gdf_column &  column) {
    typename DataType::c_type *const values =
      static_cast<typename DataType::c_type *const>(column.data);
    std::uint8_t *const valid_bits = static_cast<std::uint8_t *>(column.valid);

    static std::int64_t levels_read = 0;
    static std::int64_t values_read = 0;
    static std::int64_t nulls_count = 0;

    static const std::size_t min_batch_size = 4096;
    std::size_t              batch          = 0;
    std::size_t              batch_actual   = 0;
    std::size_t              batch_size     = 8;
    std::size_t              total_read     = 0;

    do {
        batch = ReadBatchSpaced(
          batch_size,
          definition_levels,
          repetition_levels,
          values + batch_actual,
          valid_bits + static_cast<std::ptrdiff_t>(batch_actual / 8),
          0,
          &levels_read,
          &values_read,
          &nulls_count);
        total_read += static_cast<std::size_t>(values_read);
        batch_actual += batch;
        batch_size = std::max(batch_size * 2, min_batch_size);
    } while (batch > 0 || levels_read > 0);

    return total_read;
}
#else

static inline __device__ __host__ uint8_t  _ByteWithBit(ptrdiff_t i) {
    static uint8_t values[8] =  {1, 2, 4, 8, 16, 32, 64, 128};
    return values[i];
}

static inline __device__ __host__ void _TurnBitOn(uint8_t *const bits, std::ptrdiff_t i) {
    bits[ i / 8] |= _ByteWithBit( i % 8 );

}

static inline size_t _CeilToByteLength(size_t n) {
    return (n + 7) & ~7; 
}

static inline size_t _BytesLengthToBitmapLength(size_t n){
    return _CeilToByteLength(n) / 8;
}
 

struct bitmask_functor : public thrust::binary_function<int, int16_t, int>
{
    uint8_t *const null_bitmap_ptr;
    int16_t max_definition_level;
    
    bitmask_functor(int max_definition_level, uint8_t *const null_bitmap_ptr)
     : max_definition_level(max_definition_level),
       null_bitmap_ptr(null_bitmap_ptr)  
    {

    }
    __host__ __device__ int operator()(int index, int16_t level)
    {
        if (level == max_definition_level) {
            _TurnBitOn(null_bitmap_ptr, index);
        }
        return 0;
    }
};

// #define USING_THRUST_FOR_DEF_LEVELS 1
       
static inline size_t _GenerateNullBitmap(const int16_t *const levels, const size_t levels_length, const int16_t max_definition_level, uint8_t *const null_bitmap_ptr) {
    size_t null_count = 0;
    if (max_definition_level > 0) {
        
        #ifdef USING_GPU_FOR_DEF_LEVELS
            thrust::device_vector<int16_t> d_levels(levels, levels + levels_length);

            thrust::transform(thrust::device,
                            thrust::counting_iterator<int>(0),
                            thrust::counting_iterator<int>(levels_length),
                            d_levels.begin(),
                            thrust::make_discard_iterator(), 
                            bitmask_functor{max_definition_level, null_bitmap_ptr} 
            );
            @todo: null_count in gpu
        #else
            auto num_chars = _BytesLengthToBitmapLength(levels_length);
            uint8_t * h_null_bitmap = new uint8_t[num_chars]; 
            for (int i = 0; i < levels_length; ++i) {
                if (levels[i] == max_definition_level) {
                    _TurnBitOn(h_null_bitmap, i);
                } else {
                    null_count += 1;
                }
            }
            hipMemcpy(null_bitmap_ptr, h_null_bitmap, num_chars, hipMemcpyHostToDevice);
        #endif // ! USING_GPU_FOR_DEF_LEVELS
    } else {
        auto num_chars = _BytesLengthToBitmapLength(levels_length);
        std::cout << "num_chars for valid: " << num_chars << std::endl;
        thrust::fill(thrust::device, null_bitmap_ptr, null_bitmap_ptr + num_chars - 1, 255);
        uint8_t last_char_value = 0;
        size_t levels_length_prev = levels_length - levels_length % 8;
        std::cout << "from to: " << levels_length_prev << "->" << levels_length << std::endl;
        size_t bit_index = 0;
        for (int index = levels_length_prev; index < levels_length; ++index) {
            _TurnBitOn(&last_char_value, bit_index);
            bit_index++;
        }
        thrust::fill(thrust::device, null_bitmap_ptr + num_chars - 1, null_bitmap_ptr + num_chars, last_char_value);
    }
    return null_count;
}

//@todo
// 1. read levels using gpu_decoder
// 2. 

// expands data vector that does not contain nulls into a representation that has indeterminate values where there should be nulls
// The expansion happens in place. This assumes that the data vector is actually big enough to hold the expanded data
// A vector of int work_space needs to be allocated to hold the prefix sum.



size_t get_number_of_bytes_for_valid (size_t column_size) {
    return sizeof(gdf_valid_type) * (column_size + GDF_VALID_BITSIZE - 1) / GDF_VALID_BITSIZE;
}

gdf_valid_type * get_gdf_valid_from_device(gdf_column* column) {
    gdf_valid_type * host_valid_out;
    size_t n_bytes = get_number_of_bytes_for_valid(column->size);
    host_valid_out = new gdf_valid_type[n_bytes];
    hipMemcpy(host_valid_out,column->valid, n_bytes, hipMemcpyDeviceToHost);
    return host_valid_out;
}

std::string chartobin(gdf_valid_type c, int size/* = 8*/)
{
    std::string bin;
    bin.resize(size);
    bin[0] = 0;
    int i;
    for (i = size - 1; i >= 0; i--)
    {
        bin[i] = (c % 2) + '0';
        c /= 2;
    }
    return bin;
}

std::string gdf_valid_to_str(gdf_valid_type *valid, size_t column_size)
{
    size_t n_bytes = get_number_of_bytes_for_valid(column_size);
    std::string response;
    for (int i = 0; i < n_bytes; i++)
    {
        int length = n_bytes != i + 1 ? GDF_VALID_BITSIZE : column_size - GDF_VALID_BITSIZE * (n_bytes - 1);
        auto result = chartobin(valid[i], length);
        response += std::string(result);
    }
    return response;
}

template <typename T>
void compact_to_sparse_for_nulls(T* data, const uint8_t* valid_bits, int batch_size, int * work_space){
    thrust::device_vector<uint8_t> d_valid_bits(valid_bits, valid_bits + batch_size);

    //0 1 0 1 0 1 0 1
    //0 0 1 1 2 2 3 3 
    thrust::exclusive_scan	(thrust::device, d_valid_bits.begin(), d_valid_bits.end(), work_space);

    // 1 2 3 4 5 6 7 8 
    // 
    thrust::gather_if(thrust::device, work_space, work_space + batch_size, d_valid_bits.begin(), data, data);
}

template <class DataType>
size_t ColumnReader<DataType>::ToGdfColumn(std::int16_t *const definition_levels, std::int16_t *const repetition_levels,
                                           const gdf_column &column) {

    this->HasNext();
        // num_buffered_values_ += kStep; 
    size_t values_to_read = num_buffered_values_ - num_decoded_values_; // tamanho de la pagina? 
    
    int64_t values_read;
    std::cout << "*values_to_read: " << values_to_read << std::endl;

    int64_t rows_read_total = 0;

    int16_t *levels = new int16_t[values_to_read]; // values_to_read != rows_read_total?? check this!!
    while (this->HasNext()) {
        // int16_t *levels = new int16_t[values_to_read]; // values_to_read != rows_read_total?? check this!!
        int64_t rows_read = this->ReadBatch(static_cast<std::int64_t>(values_to_read),
              levels,
              nullptr,
              static_cast<T *>(column.data + rows_read_total),
              &values_read);
        // rowgroup:  
        // accumulator = gdf_valid_concat(_GenerateNullBitmap (accumulator), _GenerateNullBitmap(current) ); 
        std::cout << "\t#rows_read: " << rows_read << std::endl;
        rows_read_total += rows_read;
    }
    std::cout << "*rows_read_total: " << rows_read_total << std::endl;
    std::cout << "*values_read: " << values_read << std::endl;
    
    auto null_count = _GenerateNullBitmap(levels, rows_read_total, descr_->max_definition_level(), column.valid);
    std::cout << "null_count: " << null_count << std::endl;

    if (rows_read_total != values_read) {
        // @todo: expand column.data to contain values_to_read
        // int* work_space;
        // hipMalloc(&work_space, sizeof(int) * rows_read_total);
        // auto valid_array = gdf_valid_to_str ( get_gdf_valid_from_device ((gdf_column*)&column), rows_read_total  );

        // compact_to_sparse_for_nulls(static_cast<T *>(column.data), (const uint8_t*)(valid_array.data()), rows_read_total, work_space);
        // hipFree(work_space);
        // values_read = values_to_read;
    }
    return static_cast<std::size_t>(values_read);
}
#endif

template class ColumnReader<::parquet::BooleanType>;
template class ColumnReader<::parquet::Int32Type>;
template class ColumnReader<::parquet::Int64Type>;
// template class ColumnReader<::parquet::Int96Type>;
template class ColumnReader<::parquet::FloatType>;
template class ColumnReader<::parquet::DoubleType>;
// template class ColumnReader<::parquet::ByteArrayType>;
// template class ColumnReader<::parquet::FLBAType>;

}  // namespace parquet
}  // namespace gdf
