#include "hip/hip_runtime.h"
/*
 * Copyright 2018 BlazingDB, Inc.
 *     Copyright 2018 Cristhian Alberto Gonzales Castillo <cristhian@blazingdb.com>
 *     Copyright 2018 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <arrow/util/bit-util.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include "column_reader.h"
#include "dictionary_decoder.cuh"
#include "plain_decoder.cuh"

namespace gdf
{
namespace parquet
{

template <class DataType, class DecoderType>
static inline void
_ConfigureDictionary(
    const ::parquet::Page *page,
    std::unordered_map<int, std::shared_ptr<DecoderType>> &decoders,
    const ::parquet::ColumnDescriptor *const column_descriptor,
    ::arrow::MemoryPool *const pool,
    DecoderType **out_decoder)
{
    const ::parquet::DictionaryPage *dictionary_page =
        static_cast<const ::parquet::DictionaryPage *>(page);

    int encoding = static_cast<int>(dictionary_page->encoding());
    if (dictionary_page->encoding() == ::parquet::Encoding::PLAIN_DICTIONARY || dictionary_page->encoding() == ::parquet::Encoding::PLAIN)
    {
        encoding = static_cast<int>(::parquet::Encoding::RLE_DICTIONARY);
    }

    auto it = decoders.find(encoding);
    if (it != decoders.end())
    {
        throw ::parquet::ParquetException(
            "Column cannot have more than one dictionary.");
    }

    if (dictionary_page->encoding() == ::parquet::Encoding::PLAIN_DICTIONARY || dictionary_page->encoding() == ::parquet::Encoding::PLAIN)
    {
        internal::PlainDecoder<DataType> dictionary(column_descriptor);
        dictionary.SetData(
            dictionary_page->num_values(), page->data(), page->size());

        auto decoder = std::make_shared<internal::DictionaryDecoder<
            DataType, gdf::arrow::internal::RleDecoder>>(column_descriptor, pool);
        decoder->SetDict(&dictionary);
        decoders[encoding] = decoder;
    }
    else
    {
        ::parquet::ParquetException::NYI(
            "only plain dictionary encoding has been implemented");
    }

    *out_decoder = decoders[encoding].get();
}

static inline bool
_IsDictionaryIndexEncoding(const ::parquet::Encoding::type &e)
{
    return e == ::parquet::Encoding::RLE_DICTIONARY || e == ::parquet::Encoding::PLAIN_DICTIONARY;
}

template <class DecoderType, class T>
static inline std::int64_t
_ReadValues(DecoderType *decoder, std::int64_t batch_size, T *out)
{
    std::int64_t num_decoded =
        decoder->Decode(out, static_cast<int>(batch_size));
    return num_decoded;
}

template <class DataType>
bool ColumnReader<DataType>::HasNext()
{
    if (num_buffered_values_ == 0 || num_decoded_values_ == num_buffered_values_)
    {
        if (!ReadNewPage() || num_buffered_values_ == 0)
        {
            return false;
        }
    }
    return true;
}

template <class DataType>
bool ColumnReader<DataType>::ReadNewPage()
{
    const std::uint8_t *buffer;

    for (;;)
    {
        current_page_ = pager_->NextPage();
        if (!current_page_)
        {
            return false;
        }

        if (current_page_->type() == ::parquet::PageType::DICTIONARY_PAGE)
        {
            _ConfigureDictionary<DataType>(current_page_.get(),
                                           decoders_,
                                           descr_,
                                           pool_,
                                           &current_decoder_);
            continue;
        }
        else if (current_page_->type() == ::parquet::PageType::DATA_PAGE)
        {
            const ::parquet::DataPage *page =
                static_cast<const ::parquet::DataPage *>(current_page_.get());

            num_buffered_values_ = page->num_values();
            num_decoded_values_ = 0;
            buffer = page->data();

            std::int64_t data_size = page->size();

            if (descr_->max_repetition_level() > 0)
            {
                std::int64_t rep_levels_bytes =
                    repetition_level_decoder_.SetData(
                        page->repetition_level_encoding(),
                        descr_->max_repetition_level(),
                        static_cast<int>(num_buffered_values_),
                        buffer);
                buffer += rep_levels_bytes;
                data_size -= rep_levels_bytes;
            }

            if (descr_->max_definition_level() > 0)
            {
                std::int64_t def_levels_bytes =
                    definition_level_decoder_.SetData(
                        page->definition_level_encoding(),
                        descr_->max_definition_level(),
                        static_cast<int>(num_buffered_values_),
                        buffer);
                buffer += def_levels_bytes;
                data_size -= def_levels_bytes;
            }

            ::parquet::Encoding::type encoding = page->encoding();

            if (_IsDictionaryIndexEncoding(encoding))
            {
                encoding = ::parquet::Encoding::RLE_DICTIONARY;
            }

            auto it = decoders_.find(static_cast<int>(encoding));
            if (it != decoders_.end())
            {
                if (encoding == ::parquet::Encoding::RLE_DICTIONARY)
                {
                    DCHECK(current_decoder_->encoding() == ::parquet::Encoding::RLE_DICTIONARY);
                }
                current_decoder_ = it->second.get();
            }
            else
            {
                switch (encoding)
                {
                case ::parquet::Encoding::PLAIN:
                {
                    std::shared_ptr<DecoderType> decoder(
                        new internal::PlainDecoder<DataType>(descr_));
                    decoders_[static_cast<int>(encoding)] = decoder;
                    current_decoder_ = decoder.get();
                    break;
                }
                case ::parquet::Encoding::RLE_DICTIONARY:
                    throw ::parquet::ParquetException(
                        "Dictionary page must be before data page.");

                case ::parquet::Encoding::DELTA_BINARY_PACKED:
                case ::parquet::Encoding::DELTA_LENGTH_BYTE_ARRAY:
                case ::parquet::Encoding::DELTA_BYTE_ARRAY:
                    ::parquet::ParquetException::NYI("Unsupported encoding");

                default:
                    throw ::parquet::ParquetException(
                        "Unknown encoding type.");
                }
            }
            current_decoder_->SetData(static_cast<int>(num_buffered_values_),
                                      buffer,
                                      static_cast<int>(data_size));
            return true;
        }
        else
        {
            continue;
        }
    }
    return true;
}

static inline bool
_HasSpacedValues(const ::parquet::ColumnDescriptor *descr)
{
    if (descr->max_repetition_level() > 0)
    {
        return !descr->schema_node()->is_required();
    }
    else
    {
        const ::parquet::schema::Node *node = descr->schema_node().get();
        while (node)
        {
            if (node->is_optional())
            {
                return true;
            }
            node = node->parent();
        }
        return false;
    }
}

static inline void
_DefinitionLevelsToBitmap(const std::int16_t *def_levels,
                          std::int64_t num_def_levels,
                          const std::int16_t max_definition_level,
                          const std::int16_t max_repetition_level,
                          std::int64_t *values_read,
                          std::int64_t *null_count,
                          std::uint8_t *valid_bits,
                          const std::int64_t valid_bits_offset)
{
    ::arrow::internal::BitmapWriter valid_bits_writer(
        valid_bits, valid_bits_offset, num_def_levels);

    for (std::int64_t i = 0; i < num_def_levels; ++i)
    {
        if (def_levels[i] == max_definition_level)
        {
            valid_bits_writer.Set();
        }
        else if (max_repetition_level > 0)
        {
            if (def_levels[i] == (max_definition_level - 1))
            {
                valid_bits_writer.Clear();
                *null_count += 1;
            }
            else
            {
                continue;
            }
        }
        else
        {
            if (def_levels[i] < max_definition_level)
            {
                valid_bits_writer.Clear();
                *null_count += 1;
            }
            else
            {
                throw ::parquet::ParquetException(
                    "definition level exceeds maximum");
            }
        }

        valid_bits_writer.Next();
    }
    valid_bits_writer.Finish();
    *values_read = valid_bits_writer.position();
}

template <class DecoderType, class T>
static inline std::int64_t
_ReadValuesSpaced(DecoderType *decoder,
                  std::int64_t batch_size,
                  T *out,
                  std::int64_t null_count,
                  std::uint8_t *valid_bits,
                  std::int64_t valid_bits_offset)
{
    return decoder->DecodeSpaced(out,
                                 static_cast<int>(batch_size),
                                 static_cast<int>(null_count),
                                 valid_bits,
                                 valid_bits_offset);
}

template <typename DataType>
inline std::int64_t
ColumnReader<DataType>::ReadBatchSpaced(std::int64_t batch_size,
                                        std::int16_t *definition_levels,
                                        std::int16_t *repetition_levels,
                                        T *values,
                                        std::uint8_t *valid_bits,
                                        std::int64_t valid_bits_offset, //
                                        std::int64_t *levels_read,
                                        std::int64_t *values_read,
                                        std::int64_t *nulls_count)
{
    if (!HasNext())
    {
        *levels_read = 0;
        *values_read = 0;
        *nulls_count = 0;
        return 0;
    }

    std::int64_t total_values;

    batch_size =
        std::min(batch_size, num_buffered_values_ - num_decoded_values_);

    if (descr_->max_definition_level() > 0)
    {
        std::int64_t num_def_levels =
            ReadDefinitionLevels(batch_size, definition_levels);

        if (descr_->max_repetition_level() > 0)
        {
            std::int64_t num_rep_levels =
                ReadRepetitionLevels(batch_size, repetition_levels);
            if (num_def_levels != num_rep_levels)
            {
                throw ::parquet::ParquetException(
                    "Number of decoded rep / def levels did not match");
            }
        }

        const bool has_spaced_values = _HasSpacedValues(descr_);

        std::int64_t null_count = 0;
        if (!has_spaced_values)
        {
            int values_to_read = 0;
            for (std::int64_t i = 0; i < num_def_levels; ++i)
            {
                if (definition_levels[i] == descr_->max_definition_level())
                {
                    ++values_to_read;
                }
            }
            std::cout << "*ReadBatchSpaced: before _ReadValues" << std::endl;

            total_values =
                _ReadValues(current_decoder_, values_to_read, values);
            for (std::int64_t i = 0; i < total_values; i++)
            {
                //check: valid_bits_offset + i
                ::arrow::BitUtil::SetBit(valid_bits, valid_bits_offset + i);
            }
            *values_read = total_values;
        }
        else
        {
            std::int16_t max_definition_level = descr_->max_definition_level();
            std::int16_t max_repetition_level = descr_->max_repetition_level();
            _DefinitionLevelsToBitmap(definition_levels,
                                      num_def_levels,
                                      max_definition_level,
                                      max_repetition_level,
                                      values_read,
                                      &null_count,
                                      valid_bits,
                                      valid_bits_offset);

            total_values = _ReadValuesSpaced(current_decoder_,
                                             *values_read,
                                             values,
                                             static_cast<int>(null_count),
                                             valid_bits,
                                             valid_bits_offset);
        }
        *levels_read = num_def_levels;
        *nulls_count = null_count;
    }
    else
    {
        total_values = _ReadValues(current_decoder_, batch_size, values);
        for (std::int64_t i = 0; i < total_values; i++)
        {
            ::arrow::BitUtil::SetBit(valid_bits, valid_bits_offset + i);
        }
        *nulls_count = 0;
        *levels_read = total_values;
    }

    ConsumeBufferedValues(*levels_read);

    return total_values;
}

template <class DataType>
inline std::int64_t
ColumnReader<DataType>::ReadBatch(std::int64_t batch_size,
                                  std::int16_t *def_levels,
                                  std::int16_t *rep_levels,
                                  T *values,
                                  std::int64_t *values_read)
{
    if (!HasNext())
    {
        *values_read = 0;
        return 0;
    }
    batch_size = std::min(batch_size, num_buffered_values_ - num_decoded_values_);

    std::int64_t num_def_levels = 0;
    std::int64_t num_rep_levels = 0;

    std::int64_t values_to_read = 0;

    if (descr_->max_definition_level() > 0 && def_levels)
    {
        num_def_levels = ReadDefinitionLevels(batch_size, def_levels);
        for (std::int64_t i = 0; i < num_def_levels; ++i)
        {
            if (def_levels[i] == descr_->max_definition_level())
            {
                ++values_to_read;
            }
        }
    }
    else
    {
        values_to_read = batch_size;
    }

    if (descr_->max_repetition_level() > 0 && rep_levels)
    {
        num_rep_levels = ReadRepetitionLevels(batch_size, rep_levels);
        if (def_levels && num_def_levels != num_rep_levels)
        {
            throw ::parquet::ParquetException(
                "Number of decoded rep / def levels did not match");
        }
    }

    *values_read = _ReadValues(current_decoder_, values_to_read, values);
    std::int64_t total_values = std::max(num_def_levels, *values_read);
    ConsumeBufferedValues(total_values);

    return total_values;
}

template <class DataType>
struct ParquetTraits
{
};

#define TYPE_TRAITS_FACTORY(ParquetType, GdfDType)      \
    template <>                                         \
    struct ParquetTraits<ParquetType>                   \
    {                                                   \
        static constexpr gdf_dtype gdfDType = GdfDType; \
    }

TYPE_TRAITS_FACTORY(::parquet::BooleanType, GDF_INT8);
TYPE_TRAITS_FACTORY(::parquet::Int32Type, GDF_INT32);
TYPE_TRAITS_FACTORY(::parquet::Int64Type, GDF_INT64);
TYPE_TRAITS_FACTORY(::parquet::FloatType, GDF_FLOAT32);
TYPE_TRAITS_FACTORY(::parquet::DoubleType, GDF_FLOAT64);

#undef TYPE_TRAITS_FACTORY

struct is_equal
{
    int16_t max_definition_level;

    is_equal(int16_t max_definition_level)
        : max_definition_level(max_definition_level)
    {

    }
    __host__ __device__ bool operator()(const int16_t &x)
    {
        return x == max_definition_level;
    }
};

// expands data vector that does not contain nulls into a representation that has indeterminate values where there should be nulls
// A vector of int work_space needs to be allocated to hold the map for the scatter operation. The workspace should be of size batch_size
template <typename T>
void compact_to_sparse_for_nulls(T *data_in, T *data_out, const int16_t *definition_levels, int16_t max_definition_level,
                                 int64_t batch_size, int *work_space)
{
    is_equal op(max_definition_level);
    auto out_iter = thrust::copy_if(thrust::device,
                                    thrust::counting_iterator<int>(0),
                                    thrust::counting_iterator<int>(batch_size),
                                    definition_levels,
                                    work_space,
                                    op);
    int num_not_null = out_iter - work_space;
    thrust::scatter(thrust::device, data_in, data_in + num_not_null, work_space, data_out);
}

#define WARP_BYTE 4
#define WARP_SIZE 32

__global__ void def_levels_to_valid(uint8_t* valid, const int16_t *def_levels, const  int size, const  int max_definition_level) {
    int blksz = blockDim.x * blockDim.y * blockDim.z;
    int blkid = gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;
    int blkof = blksz * blkid;
    int thdid = blkof + blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;

    uint32_t bitmask = 0;
    if (thdid < size) {
        bitmask = 1 << (thdid % WARP_SIZE);
        if (def_levels[thdid] == max_definition_level) {
            bitmask &= (1 << (thdid % WARP_SIZE));
        } else if (def_levels[thdid] < max_definition_level) {
            bitmask &= (0 << (thdid % WARP_SIZE));
        }
    }

    __syncwarp();

    for (int offset = 16; offset > 0; offset /= 2)
        bitmask += __shfl_down_sync(0xFFFFFFFF, bitmask, offset);

    if ((thdid % WARP_SIZE) == 0) {
        int index = thdid / WARP_SIZE * WARP_BYTE;
        valid[index + 0] = 0xFF & bitmask;
        valid[index + 1] = 0XFF & (bitmask >> 8);
        valid[index + 2] = 0XFF & (bitmask >> 16);
        valid[index + 3] = 0XFF & (bitmask >> 24);
    }
}

static inline  uint8_t _ByteWithBit(ptrdiff_t i)
{
    static uint8_t kBitmask[8] = {1, 2, 4, 8, 16, 32, 64, 128};
    return kBitmask[i];
}

static inline  uint8_t _FlippedBitmask(ptrdiff_t i)
{
    static uint8_t kFlippedBitmask[] = {254, 253, 251, 247, 239, 223, 191, 127};
    return kFlippedBitmask[i];

}

static inline  void _TurnBitOn(uint8_t *const bits, std::ptrdiff_t i)
{
    bits[i / 8] |= _ByteWithBit(i % 8);
}

static inline  void _TurnBitOff(uint8_t *const bits, std::ptrdiff_t i)
{
    bits[i / 8] &= _FlippedBitmask(i % 8);
}

static inline size_t _CeilToByteLength(size_t n) {
    return (n + 7) & ~7;
}

static inline size_t _BytesLengthToBitmapLength(size_t n){
    return _CeilToByteLength(n) / 8;
}

static inline void
_DefinitionLevelsToBitmap(const std::int16_t *d_def_levels,
                          std::int64_t        def_length,
                          const std::int16_t  max_definition_level,
                          std::int64_t *      null_count,
                          std::uint8_t *      d_valid_ptr,
                          const std::int64_t  valid_bits_offset) {

    if (max_definition_level > 0) {
        dim3 grid(2, 2, 2); //@todo: optimal params flor grid and blocks
        dim3 block(32, 2, 2);
        if (valid_bits_offset % 8 == 0) {
            def_levels_to_valid<<<grid, block>>>(d_valid_ptr + valid_bits_offset/8, d_def_levels, def_length, max_definition_level);
        } else {
            int left_bits_length = valid_bits_offset % 8;
            int rigth_bits_length = 8 - left_bits_length;
            uint8_t mask;
            hipMemcpy(&mask, d_valid_ptr + (valid_bits_offset/8), 1, hipMemcpyDeviceToHost);

            thrust::host_vector<int16_t> h_def_levels(rigth_bits_length);
            hipMemcpy(h_def_levels.data(), d_def_levels, rigth_bits_length * sizeof(int16_t), hipMemcpyDeviceToHost);
            for(size_t i = 0; i < h_def_levels.size(); i++) {
                if (h_def_levels[i] == max_definition_level) {
                    mask |= _ByteWithBit(i + left_bits_length);
                } else {
                    if (h_def_levels[i] < max_definition_level) {
                        mask &= _FlippedBitmask(i + left_bits_length);
                        //*null_count += 1; // @todo: null_count support
                    }
                }
            }
            hipMemcpy(d_valid_ptr + valid_bits_offset / 8, &mask, sizeof(uint8_t), hipMemcpyHostToDevice);
            def_levels_to_valid<<<grid, block>>>(d_valid_ptr + valid_bits_offset/8 + 1, d_def_levels + rigth_bits_length, def_length, max_definition_level);
        }
    } else {
        auto num_chars = _BytesLengthToBitmapLength(def_length);
        thrust::fill(thrust::device, d_valid_ptr, d_valid_ptr + num_chars - 1, 255);
        uint8_t last_char_value = 0;
        size_t levels_length_prev = def_length - def_length % 8;
        size_t bit_index = 0;
        for (int index = levels_length_prev; index < def_length; ++index) {
            _TurnBitOn(&last_char_value, bit_index);
            bit_index++;
        }
        thrust::fill(thrust::device, d_valid_ptr + num_chars - 1, d_valid_ptr + num_chars, last_char_value);
    }
}

template <class DataType>
size_t ColumnReader<DataType>::ToGdfColumn(std::int16_t *const definition_levels, std::int16_t *const repetition_levels,
                                           const gdf_column &column)
{
    using c_type = typename DataType::c_type;

    c_type *const values = static_cast<c_type *const>(column.data);
    std::uint8_t *const d_valid_bits = static_cast<std::uint8_t *>(column.valid);

    size_t values_to_read = num_buffered_values_ - num_decoded_values_;
    //TEST: min batches => size_t values_to_read = std::min<size_t>(8, num_buffered_values_ - num_decoded_values_);

    int64_t values_read;
    int64_t rows_read_total = 0;
    int64_t null_count = 0;
    int64_t values_read_counter = 0;

    while (this->HasNext()) {
        auto def_levels_curr = definition_levels + rows_read_total;

        int64_t rows_read = this->ReadBatch(static_cast<std::int64_t>(values_to_read),
                                            def_levels_curr,
                                            nullptr,
                                            static_cast<T *>(values + values_read_counter), // corregir saltos de values
                                            &values_read);

        thrust::device_vector<int16_t> d_def_levels(def_levels_curr, def_levels_curr + rows_read);

        _DefinitionLevelsToBitmap(thrust::raw_pointer_cast(d_def_levels.data()),
                                  rows_read,
                                  descr_->max_definition_level(),
                                  &null_count,
                                  d_valid_bits,
                                  rows_read_total);

        rows_read_total += rows_read;
        values_read_counter += values_read;
    }

    if (rows_read_total != values_read_counter) {
        thrust::device_vector<int> work_space_vector(rows_read_total);
        int* work_space = thrust::raw_pointer_cast(work_space_vector.data());
        thrust::device_vector<c_type> d_values_in(values, values + rows_read_total);
        thrust::device_vector<int16_t> d_levels(definition_levels, definition_levels + rows_read_total);

        compact_to_sparse_for_nulls(thrust::raw_pointer_cast(d_values_in.data()),
                                    values,
                                    thrust::raw_pointer_cast(d_levels.data()),
                                    descr_->max_definition_level(),
                                    rows_read_total,
                                    work_space);
    }
    return static_cast<std::size_t>(rows_read_total);
}

template <class DataType>
size_t
ColumnReader<DataType>::ToGdfColumn(const gdf_column &column, const std::ptrdiff_t offset) {
    if (!HasNext()) { return 0; }

    using c_type = typename DataType::c_type;

    c_type *const values = static_cast<c_type *const>(column.data) + offset;
    std::uint8_t *const d_valid_bits =
      static_cast<std::uint8_t *>(column.valid) + (offset / 8);

    size_t values_to_read = num_buffered_values_ - num_decoded_values_;

    int64_t values_read;
    int64_t rows_read_total     = 0;
    int64_t null_count          = 0;
    int64_t values_read_counter = 0;


    std::int16_t *definition_levels = new std::int16_t[values_to_read];
    std::int16_t *repetition_levels = new std::int16_t[values_to_read];

    do {
        auto def_levels_curr = definition_levels + rows_read_total;

        int64_t rows_read = this->ReadBatch(
          static_cast<std::int64_t>(values_to_read),
          def_levels_curr,
          nullptr,
          static_cast<T *>(
            values + values_read_counter),  // corregir saltos de values
          &values_read);

        thrust::device_vector<int16_t> d_def_levels(
          def_levels_curr, def_levels_curr + rows_read);

        _DefinitionLevelsToBitmap(
          thrust::raw_pointer_cast(d_def_levels.data()),
          rows_read,
          descr_->max_definition_level(),
          &null_count,
          d_valid_bits,
          rows_read_total + (offset % 8));

        rows_read_total += rows_read;
        values_read_counter += values_read;
    } while (this->HasNext());

    if (rows_read_total != values_read_counter) {
        thrust::device_vector<int> work_space_vector(rows_read_total);
        int *work_space = thrust::raw_pointer_cast(work_space_vector.data());
        thrust::device_vector<c_type>  d_values_in(values,
                                                  values + rows_read_total);
        thrust::device_vector<int16_t> d_levels(
          definition_levels, definition_levels + rows_read_total);

        compact_to_sparse_for_nulls(
          thrust::raw_pointer_cast(d_values_in.data()),
          values,
          thrust::raw_pointer_cast(d_levels.data()),
          descr_->max_definition_level(),
          rows_read_total,
          work_space);
    }

    delete[] definition_levels;
    delete[] repetition_levels;

    return static_cast<std::size_t>(rows_read_total);
}

template class ColumnReader<::parquet::BooleanType>;
template class ColumnReader<::parquet::Int32Type>;
template class ColumnReader<::parquet::Int64Type>;
template class ColumnReader<::parquet::FloatType>;
template class ColumnReader<::parquet::DoubleType>;

} // namespace parquet
} // namespace gdf
